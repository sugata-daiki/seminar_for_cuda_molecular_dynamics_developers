#include <iostream>
#include <vector>
#include <string>

#include "Simulation.cuh"

int main(int argc, char* argv[]) {

    if (argc < 2) {
        std::cerr << "[ error ]: No input file specified." << std::endl;
        exit(-1);
    }

    std::string ff_input = argv[1];

    System system;
    Simulation simulation;

    simulation.read_forcefields(ff_input);

    int N_particles = 10000;
    int num_steps = 10000;
    float temperature = 300.0; // K
    float timestep = 0.01; // ns
    int seed = 12345;
    float diffusion_coefficient = 10.0; // [(um**2)/s)]
    float r0 = 10.0;    
    float kbond = 2.5;

    simulation.set_timestep(timestep);
    simulation.set_seed(seed);
    system.set_params(temperature);

    for (int i = 0; i < N_particles; i++) {
        system.add_particle("DNA", diffusion_coefficient, i);
        simulation.set_positions(i*9.0f, 0.0f, 0.0f, 0.0f);
    }

    int numBonds = N_particles - 1;

    for (int i = 0; i < numBonds; i++) {
        system.add_bond(r0, kbond, i, i+1);
    }

    simulation.run(system, num_steps);
    
    hipDeviceSynchronize();

}
